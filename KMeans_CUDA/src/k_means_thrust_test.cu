#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/copy.h>
#include <thrust/logical.h>
#include "k_means_thrust.h"
#include "common.h"

typedef thrust::device_vector<real> dv_real;
typedef thrust::device_vector<int> dv_int;
typedef thrust::host_vector<real> hv_real;

typedef thrust::tuple<real, int> real_indexed;

#define POW2(x) ((x) * (x))

// 定義 compute_distances 核心函數
__global__ void compute_distances(
    const real* points, const real* centroids, real* distances,
    int dims, int n_points, int n_clusters) {
    
    int point_id = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (point_id < n_points) {
        for (int j = 0; j < n_clusters; j++) {
            real distance = 0.0;
            for (int d = 0; d < dims; d++) {
                real diff = points[point_id * dims + d] - centroids[j * dims + d];
                distance += diff * diff;
            }
            distances[point_id * n_clusters + j] = distance;
        }
    }
}

struct maximum_by_first : public thrust::binary_function<real_indexed, real_indexed, real_indexed> {
    maximum_by_first() {}

    __host__ __device__
    real_indexed operator()(real_indexed x_1, real_indexed x_2) {
        return thrust::get<0>(x_1) < thrust::get<0>(x_2) ? x_1 : x_2;
    }
};

#define D_PRINT_POINT(x, d, i) { \
  if(DEBUG_TEST) {\
    thrust::copy_n( \
      x.begin() + i*d, \
      d, \
      std::ostream_iterator<real>(std::cerr, ", ") \
    ); \
    std::cerr << std::endl;\
  }\
}
#define D_PRINT_ALL(x) { \
  if(DEBUG_TEST) {\
    thrust::copy( \
      x.begin(), \
      x.end(), \
      std::ostream_iterator<real>(std::cerr, ", ") \
    ); \
    std::cerr << std::endl;\
  }\
}

struct distance_component : public thrust::unary_function<int, real> {
    const int d, k;
    const real* points, * centroids;

    distance_component(int _d, int _k, real* _points, real* _centroids)
        : d(_d), k(_k), points(_points), centroids(_centroids) {}

    __host__ __device__
    real operator()(int index) {
        int i = (index / d) / k;
        int j = (index / d) % k;
        int l = index % d;
        return POW2(points[i * d + l] - centroids[j * d + l]);
    }
};

struct updateCentroidSums_thrust : public thrust::unary_function<void, thrust::tuple<real, int>> {
    const int d;
    const int* d_point_cluster_ids;
    real* centroids;

    updateCentroidSums_thrust(int _d, int* _d_point_cluster_ids, real* _centroids)
        : d(_d), d_point_cluster_ids(_d_point_cluster_ids), centroids(_centroids) {}

    __device__
    void operator()(thrust::tuple<real, int> real_index) {
        real value = thrust::get<0>(real_index);
        int index = thrust::get<1>(real_index);
        int target_centroid_id = d_point_cluster_ids[index / d];
        int target_centroid_component_id = target_centroid_id * d + index % d;
        atomicAdd(centroids + target_centroid_component_id, value);
    }
};

struct computeCentroidAverages_thrust : public thrust::unary_function<void, thrust::tuple<int, int>> {
    const int d, i;
    real* centroids;

    computeCentroidAverages_thrust(int _d, int _i, real* _centroids)
        : d(_d), i(_i), centroids(_centroids) {}

    __device__
    void operator()(thrust::tuple<int, int> ints) {
        int centroid_id = thrust::get<0>(ints);
        int counts = thrust::get<1>(ints);
        *(centroids + centroid_id * d + i) /= counts;
    }
};

struct l1_op : public thrust::unary_function<bool, thrust::tuple<real, real>> {
    const real l1_thresh;
    l1_op(real _l1_thresh) : l1_thresh(_l1_thresh) {}

    __host__ __device__
    bool operator()(thrust::tuple<real, real> realz) {
        real x1 = thrust::get<0>(realz);
        real x2 = thrust::get<1>(realz);
        return abs(x1 - x2) < l1_thresh;
    }
};

int k_means_thrust_test(int n_points, real* data_points, struct options_t opts, int* point_cluster_ids, real* centroids, double& per_iteration_time) {
    using namespace thrust;
    using namespace thrust::placeholders;

    bool done = false;
    int iterations = 0;

    // CUDA events for timing
    hipEvent_t start_total, stop_total, start_transfer, stop_transfer;
    hipEvent_t start_iter, stop_iter;
    float transfer_time = 0.0, total_runtime = 0.0;
    hipEventCreate(&start_total);
    hipEventCreate(&stop_total);
    hipEventCreate(&start_transfer);
    hipEventCreate(&stop_transfer);
    hipEventCreate(&start_iter);
    hipEventCreate(&stop_iter);

    hipEventRecord(start_total, 0);  // Start overall timing

    // 設置 block 和 thread 的大小
    int threadsPerBlock = 1024;
    int blocksPerGrid = (n_points + threadsPerBlock - 1) / threadsPerBlock;

    // 將資料從主機（CPU）傳輸到設備（GPU） (Step 1)
    hipEventRecord(start_transfer, 0);
    dv_real d_points(data_points, data_points + n_points * opts.dimensions);
    dv_real old_centroids(centroids, centroids + opts.n_clusters * opts.dimensions);

    hipEventRecord(stop_transfer, 0);
    hipEventSynchronize(stop_transfer);

    // 計算從主機到設備的傳輸時間
    float host_to_device_transfer_time;
    hipEventElapsedTime(&host_to_device_transfer_time, start_transfer, stop_transfer);
    transfer_time += host_to_device_transfer_time;

    // Create variables for computation
    dv_real new_centroids(opts.n_clusters * opts.dimensions);
    dv_real point_centroid_distances(n_points * opts.n_clusters);
    dv_int d_point_cluster_ids(n_points);
    dv_int point_to_centroid_map(n_points);
    dv_int d_k_counts(opts.n_clusters);
    dv_int d_k_count_keys(opts.n_clusters);

    // Step 2: K-means iteration loop
    while (!done) {
        hipEventRecord(start_iter, 0);

        // Compute distance components (Step 2.1)
        distance_component dist_comp(opts.dimensions, opts.n_clusters, raw_pointer_cast(d_points.data()), raw_pointer_cast(old_centroids.data()));

        // Launch custom kernel for computing distances
        compute_distances<<<blocksPerGrid, threadsPerBlock>>>(
            raw_pointer_cast(d_points.data()), raw_pointer_cast(old_centroids.data()),
            raw_pointer_cast(point_centroid_distances.data()), opts.dimensions, n_points, opts.n_clusters
        );

        if (PERFORMANCE_TEST) {
            hipEventRecord(stop_iter, 0);
            hipEventSynchronize(stop_iter);
            float step_time;
            hipEventElapsedTime(&step_time, start_iter, stop_iter);
            printf("Step 2.1 (Distance computation): %f ms\n", step_time);
            hipEventRecord(start_iter);
        }

        // Assign points to centroids (Step 2.2)
        reduce_by_key(
            thrust::cuda::par.on(0),
            make_transform_iterator(counting_iterator<int>(0), _1 / opts.n_clusters),
            make_transform_iterator(counting_iterator<int>(n_points * opts.n_clusters), _1 / opts.n_clusters),
            make_zip_iterator(make_tuple(point_centroid_distances.begin(), counting_iterator<int>(0))),
            make_discard_iterator(),
            make_zip_iterator(make_tuple(make_discard_iterator(), point_to_centroid_map.begin())),
            equal_to<int>(),
            maximum_by_first()
        );

        transform(
            thrust::cuda::par.on(0),
            point_to_centroid_map.begin(),
            point_to_centroid_map.end(),
            point_to_centroid_map.begin(),
            _1 % opts.n_clusters
        );

        if (PERFORMANCE_TEST) {
            hipEventRecord(stop_iter, 0);
            hipEventSynchronize(stop_iter);
            float step_time;
            hipEventElapsedTime(&step_time, start_iter, stop_iter);
            printf("Step 2.2 (Assign points to centroids): %f ms\n", step_time);
            hipEventRecord(start_iter);
        }

        fill(thrust::cuda::par.on(0), new_centroids.begin(), new_centroids.end(), 0);
        updateCentroidSums_thrust compute_assign_means(opts.dimensions, raw_pointer_cast(point_to_centroid_map.data()), raw_pointer_cast(new_centroids.data()));
        for_each(
            thrust::cuda::par.on(0),
            make_zip_iterator(make_tuple(d_points.begin(), counting_iterator<int>(0))),
            make_zip_iterator(make_tuple(d_points.end(), counting_iterator<int>(n_points * opts.dimensions))),
            compute_assign_means
        );

        if (PERFORMANCE_TEST) {
            hipEventRecord(stop_iter, 0);
            hipEventSynchronize(stop_iter);
            float step_time;
            hipEventElapsedTime(&step_time, start_iter, stop_iter);
            printf("Step 2.3 (Compute new centroids): %f ms\n", step_time);
            hipEventRecord(start_iter);
        }

        d_point_cluster_ids = point_to_centroid_map;
        sort(thrust::cuda::par.on(0), d_point_cluster_ids.begin(), d_point_cluster_ids.end());
        auto new_end = reduce_by_key(
            thrust::cuda::par.on(0),
            d_point_cluster_ids.begin(),
            d_point_cluster_ids.end(),
            make_constant_iterator(1),
            d_k_count_keys.begin(),
            d_k_counts.begin()
        );

        fill(thrust::cuda::par.on(0), new_end.second, d_k_counts.end(), 0);

        if (PERFORMANCE_TEST) {
            hipEventRecord(stop_iter, 0);
            hipEventSynchronize(stop_iter);
            float step_time;
            hipEventElapsedTime(&step_time, start_iter, stop_iter);
            printf("Step 2.4 (Sort cluster IDs and reduce): %f ms\n", step_time);
            hipEventRecord(start_iter);
        }

        for (int i = 0; i < opts.dimensions; i++) {
            for_each(
                thrust::cuda::par.on(0),
                make_zip_iterator(make_tuple(d_k_count_keys.begin(), d_k_counts.begin())),
                make_zip_iterator(make_tuple(new_end.first, new_end.second)),
                computeCentroidAverages_thrust(opts.dimensions, i, raw_pointer_cast(new_centroids.data()))
            );
        }

        if (PERFORMANCE_TEST) {
            hipEventRecord(stop_iter, 0);
            hipEventSynchronize(stop_iter);
            float step_time;
            hipEventElapsedTime(&step_time, start_iter, stop_iter);
            printf("Step 2.5 (Divide centroids by counts): %f ms\n", step_time);
        }

        swap(new_centroids, old_centroids);
        real l1_thresh = opts.threshold / opts.dimensions;
        bool converged = transform_reduce(
            thrust::cuda::par.on(0),
            make_zip_iterator(make_tuple(new_centroids.begin(), old_centroids.begin())),
            make_zip_iterator(make_tuple(new_centroids.end(), old_centroids.end())),
            l1_op(l1_thresh),
            true,
            logical_and<bool>()
        );

        iterations++;
        done = (iterations > opts.max_iterations) || converged;
    }

    hipEventRecord(start_transfer, 0);
    copy(thrust::cuda::par.on(0), old_centroids.begin(), old_centroids.end(), centroids);
    copy(thrust::cuda::par.on(0), point_to_centroid_map.begin(), point_to_centroid_map.end(), point_cluster_ids);
    hipEventRecord(stop_transfer, 0);
    hipEventSynchronize(stop_transfer);

    float device_to_host_transfer_time;
    hipEventElapsedTime(&device_to_host_transfer_time, start_transfer, stop_transfer);
    transfer_time += device_to_host_transfer_time;

    hipEventRecord(stop_total, 0);
    hipEventSynchronize(stop_total);
    hipEventElapsedTime(&total_runtime, start_total, stop_total);
    per_iteration_time = total_runtime / iterations;

    hipEventDestroy(start_transfer);
    hipEventDestroy(stop_transfer);
    hipEventDestroy(start_total);
    hipEventDestroy(stop_total);
    hipEventDestroy(start_iter);
    hipEventDestroy(stop_iter);

    if (opts.debug || PERFORMANCE_TEST) {
        std::cout << "Total runtime: " << total_runtime << " ms" << std::endl;
        std::cout << "Total data transfer time: " << transfer_time << " ms" << std::endl;
        std::cout << "Fraction of time spent on data transfer: " << (transfer_time / total_runtime) << std::endl;
    }
    print_ExecuteTime(iterations, per_iteration_time,total_runtime);
    return iterations;
}
