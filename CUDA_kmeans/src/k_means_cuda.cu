#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cfloat>  
#include "common.h"
#include "argparse.h"  
#include "io.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cmath>

// 定義一個檢查 CUDA 調用是否成功
#define CHECK_ERROR(call) {                                                           \
    const hipError_t error = call;                                             \
    if (error != hipSuccess) {                                                 \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                           \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error));      \
        exit(1);                                                                \
    }                                                                           \
}

// 檢查是否所有聚類都已經收斂 Check if all clusters have converged
bool check_convergence(double* convergence_k, double threshold, int n_clusters) {
    bool converge_result = true;
    for (int i = 0; i < n_clusters; i++) {
        if (convergence_k[i] > threshold) {  
            converge_result = false;
            return converge_result;
        }
    }
    return converge_result;
}

// 定義計算歐式距離的函數 Define a function that calculates Euclidean distance
__device__ double euclideanDistance_cuda(int dims, double *point, double *centroid) {
    double sum = 0.0;
    for (int i = 0; i < dims; i++) {
        sum += (point[i] - centroid[i]) * (point[i] - centroid[i]); 
    }
    return sqrt(sum); 
}

// 找到每個點最近的聚類質心 Find the nearest cluster centroid for each point
__global__ void findNearestCentroids_Cuda(double* points, double* centroids, int* labels, int dims, int n_clusters, int num_points) {
    int point_label_index = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (point_label_index < num_points) {
        int point_index = point_label_index * dims;
        int nearest_centroid = -1;
        double min_distance = DBL_MAX;

        // 對每個質心計算歐式距離
        //centroid_0: [x0, y0, z0] -> indices 0, 1, 2
        //centroid_1: [x1, y1, z1] -> indices 3, 4, 5
        //centroid_2: [x2, y2, z2] -> indices 6, 7, 8
        //centroid_3: [x3, y3, z3] -> indices 9, 10, 11
        for (int j = 0; j < n_clusters; j++) {
            double distance = euclideanDistance_cuda(dims, &points[point_index], &centroids[j * dims]);  // 呼叫 CUDA 版本的歐式距離計算函數
            
            if (distance < min_distance) {
                min_distance = distance;
                nearest_centroid = j;
            }
        }

        labels[point_label_index] = nearest_centroid;  // 記錄最接近的質心標籤
    }
}

// 將每個點的坐標加到對應質心的累加和中，並增加質心的點數 Add the coordinates of each point to the accumulated sum of the corresponding centroid
__global__ void updateCentroidSums_cuda(double* points, double* centroids, int* labels, int* cluster_label_count, int dims, int num_points) {
    int point_id = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (point_id >= num_points) {
        return;
    }
    int cluster_id = labels[point_id];
    int points_start_index = point_id * dims;
    int center_start_index = cluster_id * dims;

    // 將每個點的坐標加到對應質心中
    for (int d = 0; d < dims; d++) {
        atomicAdd(&centroids[center_start_index + d], points[points_start_index + d]);
    }
    atomicAdd(&cluster_label_count[cluster_id], 1);  // 增加對應質心的點數  
}

// 核計算每個質心的平均值（將累加值除以點的數量）Calculate the average of each centroid
__global__ void averageLabeledCentroids_cuda(double* centroids, int* cluster_label_count, int dims, int n_clusters) {
    int cluster_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (cluster_id >= n_clusters) {
        return;
    }
    int cluster_start_index = cluster_id * dims;
    for (int d = 0; d < dims; d++) {
        centroids[cluster_start_index + d] = centroids[cluster_start_index + d] / cluster_label_count[cluster_id];
    }
}

__global__ void convergence_Cuda(double* centroids, double* old_centroids, double* convergence_k, int dims, int n_clusters) {
    int cluster_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (cluster_id >= n_clusters) {
        return;
    }
    
    double cluster_converge_sum = 0;
    int ci_start = cluster_id * dims;
    for (int d = 0; d < dims; d++) {
        cluster_converge_sum += fabs((centroids[ci_start + d] - old_centroids[ci_start + d]) / old_centroids[ci_start + d] * 100);
    }
    convergence_k[cluster_id] = cluster_converge_sum;
}

int k_means_cuda(int n_points, real* data_points, struct options_t opts, int* point_cluster_ids, real* centroids, double& per_iteration_time) {
    // 計算每個數組所需的記憶體大小
    size_t points_size = n_points * opts.dimensions * sizeof(real); 
    size_t centroids_size = opts.n_clusters * opts.dimensions * sizeof(real); 
    size_t labels_size = n_points * sizeof(int); 
    size_t cluster_count_size = opts.n_clusters * sizeof(int);  
    size_t convergence_size = opts.n_clusters * sizeof(double);  

    // 定義GPU上的變量，這些變量將被分配記憶體
    real *d_points, *d_centroids, *d_old_centroids, *d_convergence_k;
    int *d_point_cluster_label, *d_cluster_label_count;
    double* convergence_k = new double[opts.n_clusters] { 0 };

    // 計時器初始化
    hipEvent_t start_transfer, stop_transfer, start_total, stop_total, start_iter, stop_iter;
    float transfer_time = 0.0, total_runtime = 0.0;

    // 初始化收斂標誌和迭代計數器
    bool converged = false;
    int iterations = 0;
    float time_step;

    // Create CUDA events for timing
    hipEventCreate(&start_transfer);
    hipEventCreate(&stop_transfer);
    hipEventCreate(&start_total);
    hipEventCreate(&stop_total);
    hipEventCreate(&start_iter);
    hipEventCreate(&stop_iter);

    // Start timing total runtime
    hipEventRecord(start_total, 0);

    // 在GPU上分配記憶體
    CHECK_ERROR(hipMalloc((void**) &d_points, points_size)); 
    CHECK_ERROR(hipMalloc((void**) &d_centroids, centroids_size)); 
    CHECK_ERROR(hipMalloc((void**) &d_point_cluster_label, labels_size));  
    CHECK_ERROR(hipMalloc((void**) &d_cluster_label_count, cluster_count_size)); 
    CHECK_ERROR(hipMalloc((void**) &d_convergence_k, convergence_size)); 
    CHECK_ERROR(hipMalloc((void**) &d_old_centroids, centroids_size)); 

    // 計算從主機到設備的傳輸時間 Calculate transfer time from host to device
    //hipError_t hipMemcpy(void *dst, const void *src, size_t count, hipMemcpyKind kind);
    hipEventRecord(start_transfer, 0);
    CHECK_ERROR(hipMemcpy(d_points, data_points, points_size, hipMemcpyHostToDevice));  
    CHECK_ERROR(hipMemcpy(d_centroids, centroids, centroids_size, hipMemcpyHostToDevice));  
    hipEventRecord(stop_transfer, 0);
    hipEventSynchronize(stop_transfer);
    
    // 計算從主機到設備的傳輸時間
    float device_to_device_transfer_time=0.0;
    float host_to_device_transfer_time;
    hipEventElapsedTime(&host_to_device_transfer_time, start_transfer, stop_transfer);
    transfer_time += host_to_device_transfer_time;

    // 設置每個block的執行線程數以及block的總數
    int threadsPerBlock = 256;  // 每個block的線程數
    int n_blocksPerGrid = (n_points + threadsPerBlock - 1) / threadsPerBlock;  // 每個點對應的block數量 (N+block size -1)/block size
    int n_blocks_cluster = (opts.n_clusters + threadsPerBlock - 1) / threadsPerBlock;  // 每個聚類對應的block數量

    if(opts.debug or PERFORMANCE_TEST) {
        printf("Step 0 threadsPerBlock_PerBlock: %d\n", threadsPerBlock);           // Use %d for int
        printf("Step 0 blocks points: %d\n", n_blocksPerGrid);           // Use %d for int
        printf("Step 0 cluster by block: %d\n", n_blocks_cluster); // Use %d for int
        printf("Step 0 data points: %d\n", n_points);        // Use %zu for size_t
    }

    // 開始K-means迭代，直到達到最大迭代次數或收斂
    while (!converged && iterations < opts.max_iterations) {
        // 開始計時每次迭代
        hipEventRecord(start_iter, 0);
        iterations++;  // 每次進行迭代時增加計數

        // 1. 更新每個資料點的最近質心標籤 Calculating Euclidean Distance and Assigning data points to closest centroids
        findNearestCentroids_Cuda<<<n_blocksPerGrid, threadsPerBlock>>>(d_points, d_centroids, d_point_cluster_label, opts.dimensions, opts.n_clusters, n_points);
        CHECK_ERROR(hipDeviceSynchronize());  // 確保核函數執行完成
        
        if(PERFORMANCE_TEST) {
            hipEventRecord(stop_iter);  
            hipEventSynchronize(stop_iter);
            hipEventElapsedTime(&time_step, start_iter, stop_iter);
            printf("Step 1 (findNearestCentroids): %f ms\n", time_step);
            hipEventRecord(start_iter);
        }

        // 2. 保存舊的質心以便於收斂檢查 Copy centroids_new back to centroids_old
        CHECK_ERROR(hipMemcpy(d_old_centroids, d_centroids, centroids_size, hipMemcpyDeviceToDevice));

        if(PERFORMANCE_TEST) {
            hipEventRecord(stop_iter);  
            hipEventSynchronize(stop_iter);
            hipEventElapsedTime(&time_step, start_iter, stop_iter);
            printf("Step 1 (hipMemcpyDeviceToDevice): %f ms\n", time_step);
            device_to_device_transfer_time+=time_step;
            hipEventRecord(start_iter);
        }        

        // 3. 重置新的質心和每個聚類的點數統計
        hipMemset(d_centroids, 0, centroids_size);  // 清空質心
        hipMemset(d_cluster_label_count, 0, cluster_count_size);  // 清空聚類點數

        // 4. 累加點到各個質心的總和 Compute new centroids
        updateCentroidSums_cuda<<<n_blocksPerGrid, threadsPerBlock>>>(d_points, d_centroids, d_point_cluster_label, d_cluster_label_count, opts.dimensions, n_points);
        CHECK_ERROR(hipDeviceSynchronize());
        
        if(PERFORMANCE_TEST) {
            hipEventRecord(stop_iter);  
            hipEventSynchronize(stop_iter);
            hipEventElapsedTime(&time_step, start_iter, stop_iter);
            printf("Step 2 (updateCentroidSums_cuda): %f ms\n", time_step);
            hipEventRecord(start_iter);
        }

        // 5. 計算每個聚類的平均質心 Compute new centroids
        averageLabeledCentroids_cuda<<<n_blocks_cluster, threadsPerBlock>>>(d_centroids, d_cluster_label_count, opts.dimensions, opts.n_clusters);
        CHECK_ERROR(hipDeviceSynchronize());

        // 6. 計算質心的收斂程度 Check for convergence using opts.threshold as epsilon
        hipMemset(d_convergence_k, 0, convergence_size);  // 初始化收斂數據
        convergence_Cuda<<<n_blocks_cluster, threadsPerBlock>>>(d_centroids, d_old_centroids, d_convergence_k, opts.dimensions, opts.n_clusters);
        CHECK_ERROR(hipDeviceSynchronize());

        // 7. 檢查是否達到收斂條件
        hipEventRecord(start_iter);
        CHECK_ERROR(hipMemcpy(convergence_k, d_convergence_k, convergence_size, hipMemcpyDeviceToHost));

        if(PERFORMANCE_TEST) {
            hipEventRecord(stop_iter);  
            hipEventSynchronize(stop_iter);
            hipEventElapsedTime(&time_step, start_iter, stop_iter);
            printf("Step 3 (hipMemcpyDeviceToDevice): %f ms\n", time_step);
            device_to_device_transfer_time+=time_step;
            hipEventRecord(start_iter);
        }       


        converged = check_convergence(convergence_k, opts.threshold, opts.n_clusters); 

        // 結束本次迭代的計時
        hipEventRecord(stop_iter, 0);
        hipEventSynchronize(stop_iter);

        if(PERFORMANCE_TEST) {
            // 打印聚類的點數統計
            //int* h_cluster_label_count = (int*)malloc(cluster_count_size);
            //CHECK_ERROR(hipMemcpy(h_cluster_label_count, d_cluster_label_count, cluster_count_size, hipMemcpyDeviceToHost));
            //print_ClusterLabelCount(h_cluster_label_count, opts.n_clusters);
            //free(h_cluster_label_count);  

            // 將質心從 GPU 拷貝回主機並打印
            //real* h_centroids = (real*)malloc(centroids_size);
            //CHECK_ERROR(hipMemcpy(h_centroids, d_centroids, centroids_size, hipMemcpyDeviceToHost));
            //print_Centers(h_centroids, opts.n_clusters, opts.dimensions);
            //free(h_centroids);  // 釋放主機上的記憶體
            hipEventElapsedTime(&time_step, start_iter, stop_iter);
            printf("Step 3 (convergence_Cuda): %f ms\n", time_step);
        }

        // 計算本次迭代的時間並累加
        //hipEventElapsedTime(&iteration_time, start_iter, stop_iter);
        //time_taken += iteration_time;
    }

    // Measure time for data transfer back to CPU
    hipEventRecord(start_transfer, 0);
    CHECK_ERROR(hipMemcpy(point_cluster_ids, d_point_cluster_label, labels_size, hipMemcpyDeviceToHost));  // 標籤結果
    CHECK_ERROR(hipMemcpy(centroids, d_centroids, centroids_size, hipMemcpyDeviceToHost));  // 最終質心結果
    hipEventRecord(stop_transfer, 0);
    hipEventSynchronize(stop_transfer);

    // 計算從設備到主機的傳輸時間 Calculate transfer time from device to host
    float device_to_host_transfer_time;
    hipEventElapsedTime(&device_to_host_transfer_time, start_transfer, stop_transfer);
    transfer_time += device_to_host_transfer_time;

    // Record the end of the total runtime
    hipEventRecord(stop_total, 0);
    hipEventSynchronize(stop_total);

    // Calculate total runtime
    hipEventElapsedTime(&total_runtime, start_total, stop_total);
    per_iteration_time = total_runtime / iterations;
    float time_taken = total_runtime - transfer_time;

    // Cleanup memory and events
    hipFree(d_points);
    hipFree(d_centroids);
    hipFree(d_old_centroids);
    hipFree(d_convergence_k);
    hipFree(d_point_cluster_label);
    hipFree(d_cluster_label_count);
    hipFree(convergence_k);


    // Cleanup events
    hipEventDestroy(start_transfer);
    hipEventDestroy(stop_transfer);
    hipEventDestroy(start_total);
    hipEventDestroy(stop_total);
    hipEventDestroy(start_iter);
    hipEventDestroy(stop_iter);

    // Print total runtime and transfer time information
    if(opts.debug or PERFORMANCE_TEST) {
        std::cout << "Host to Device transfer time: " << host_to_device_transfer_time << " ms" << std::endl;
        std::cout << "Device to Device transfer time: " << device_to_device_transfer_time << " ms" << std::endl;
        std::cout << "Device to Host transfer time: " << device_to_host_transfer_time << " ms" << std::endl;
        std::cout << "Total runtime: " << total_runtime << " ms" << std::endl;
        std::cout << "Total data transfer time: " << transfer_time << " ms" << std::endl;
        std::cout << "Fraction of time spent on data transfer: " << (transfer_time / total_runtime) << std::endl;
        std::cout << "Algorithm Time taken by function: " << time_taken << " ms" << std::endl;
    }

    // Return average iteration time
    per_iteration_time = time_taken / iterations;
    print_ExecuteTime(iterations, per_iteration_time,time_taken);
    return iterations;
}

