#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/reduce.h>
#include <thrust/copy.h>
#include <thrust/logical.h>
#include "k_means_thrust.h"
#include "common.h"

typedef thrust::device_vector<real> dv_real;
typedef thrust::device_vector<int> dv_int;
typedef thrust::host_vector<real> hv_real;

typedef thrust::tuple<real, int> real_indexed; // 新增 real_indexed 定義

struct maximum_by_first : public thrust::binary_function<real_indexed, real_indexed, real_indexed> {
    maximum_by_first() {}

    __host__ __device__
    real_indexed operator()(real_indexed x_1, real_indexed x_2) {
        return thrust::get<0>(x_1) < thrust::get<0>(x_2) ? x_1 : x_2;
    }
};

#define D_PRINT_POINT(x, d, i) { \
  if(DEBUG_TEST) {\
    thrust::copy_n( \
      x.begin() + i*d, \
      d, \
      std::ostream_iterator<real>(std::cerr, ", ") \
    ); \
    std::cerr << std::endl;\
  }\
}
#define D_PRINT_ALL(x) { \
  if(DEBUG_TEST) {\
    thrust::copy( \
      x.begin(), \
      x.end(), \
      std::ostream_iterator<real>(std::cerr, ", ") \
    ); \
    std::cerr << std::endl;\
  }\
}

// 定義計算歐式距離的函數 Define a function that calculates Euclidean distance
struct euclideanDistance : public thrust::unary_function<int, real> {
    const int d, k;
    const real* points, * centroids;

    euclideanDistance(int _d, int _k, real* _points, real* _centroids)
        : d(_d), k(_k), points(_points), centroids(_centroids) {}

    __host__ __device__
    real operator()(int index) {
        int i = (index / d) / k;
        int j = (index / d) % k;
        int l = index % d;
        return (points[i * d + l] - centroids[j * d + l]) * (points[i * d + l] - centroids[j * d + l]);
    }
};

// 將每個點的坐標加到對應質心的累加和中，並增加質心的點數 Add the coordinates of each point to the accumulated sum of the corresponding centroid
struct updateCentroidSums_thrust_op : public thrust::unary_function<void, thrust::tuple<real, int>> {
    const int d;
    const int* d_point_cluster_ids;
    real* centroids;

    updateCentroidSums_thrust_op(int _d, int* _d_point_cluster_ids, real* _centroids)
        : d(_d), d_point_cluster_ids(_d_point_cluster_ids), centroids(_centroids) {}

    __device__
    void operator()(thrust::tuple<real, int> real_index) {
        real value = thrust::get<0>(real_index);
        int index = thrust::get<1>(real_index);
        int target_centroid_id = d_point_cluster_ids[index / d];
        int target_centroid_component_id = target_centroid_id * d + index % d;
        atomicAdd(centroids + target_centroid_component_id, value);
    }
};

// 核計算每個質心的平均值（將累加值除以點的數量）Calculate the average of each centroid
struct averageLabeledCentroids_thrust_op : public thrust::unary_function<void, thrust::tuple<int, int>> {
    const int d, i;
    real* centroids;

    averageLabeledCentroids_thrust_op(int _d, int _i, real* _centroids)
        : d(_d), i(_i), centroids(_centroids) {}

    __device__
    void operator()(thrust::tuple<int, int> ints) {
        int centroid_id = thrust::get<0>(ints);
        int counts = thrust::get<1>(ints);
        *(centroids + centroid_id * d + i) /= counts;
    }
};

struct check_convergence_thrust_op : public thrust::unary_function<bool, thrust::tuple<real, real>> {
    const real threshold;
    check_convergence_thrust_op(real _threshold) : threshold(_threshold) {}

    __host__ __device__
    bool operator()(thrust::tuple<real, real> realz) {
        real x1 = thrust::get<0>(realz);
        real x2 = thrust::get<1>(realz);
        return abs(x1 - x2) < threshold;
    }
};

int k_means_thrust_optimized(int n_points, real* data_points, struct options_t opts, int* point_cluster_ids, real* centroids, double& per_iteration_time) {
    using namespace thrust;
    using namespace thrust::placeholders;

    bool done = false;
    int iterations = 0;
    //int k = opts.n_clusters;
    //int d = opts.dimensions;
    float time_step;

    // CUDA events for timing
    hipEvent_t start_total, stop_total, start_transfer, stop_transfer;
    hipEvent_t start_iter, stop_iter;  // Events for step timing
    float transfer_time = 0.0, total_runtime = 0.0;
    hipEventCreate(&start_total);
    hipEventCreate(&stop_total);
    hipEventCreate(&start_transfer);
    hipEventCreate(&stop_transfer);
    hipEventCreate(&start_iter);
    hipEventCreate(&stop_iter);

    hipEventRecord(start_total, 0);  // Start overall timing

    // 將資料從主機（CPU）傳輸到設備（GPU） (Step 1)
    hipEventRecord(start_transfer, 0);
    dv_real d_points(data_points, data_points + n_points * opts.dimensions);
    dv_real old_centroids(centroids, centroids + opts.n_clusters * opts.dimensions);

    hipEventRecord(stop_transfer, 0);
    hipEventSynchronize(stop_transfer);

    // 計算從主機到設備的傳輸時間 Calculate transfer time from host to device
    float host_to_device_transfer_time;
    hipEventElapsedTime(&host_to_device_transfer_time, start_transfer, stop_transfer);
    transfer_time += host_to_device_transfer_time;

    // Create variables for computation
    dv_real new_centroids(opts.n_clusters * opts.dimensions);
    dv_real point_centroid_distances(n_points * opts.n_clusters);
    dv_int d_point_cluster_ids(n_points);
    dv_int point_to_centroid_map(n_points);
    dv_int d_k_counts(opts.n_clusters);
    dv_int d_k_count_keys(opts.n_clusters);

    // Step 2: K-means iteration loop
    // 開始K-means迭代，直到達到最大迭代次數或收斂
    while (!done) {
        // 開始計時每次迭代
        hipEventRecord(start_iter, 0);  // Start timing for this iteration step

        //DEBUG_OUT("Old centroids:");
        //D_PRINT_ALL(old_centroids);

        // Compute distance components (Step 2.1)
        // 1. 更新每個資料點的最近質心標籤
        // Map each point to its nearest centroid
        euclideanDistance dist_comp(opts.dimensions, opts.n_clusters, raw_pointer_cast(d_points.data()), raw_pointer_cast(old_centroids.data()));

        reduce_by_key(
            make_transform_iterator(counting_iterator<int>(0), _1 / opts.dimensions), // Key input start: Transforms the index to represent a cluster based on the dimensional index
            make_transform_iterator(counting_iterator<int>(n_points * opts.n_clusters * opts.dimensions), _1 / opts.dimensions), // Key input end: Same transformation for the range end
            make_transform_iterator(counting_iterator<int>(0), dist_comp), // Value input: Computes some comparison metric using 'dist_comp' for each transformed index
            make_discard_iterator(), // Output key: Discards the output keys (not used)
            point_centroid_distances.begin() // Output value: Stores the results of the reduction (comparison values) in 'point_centroid_distances'
        );

        if (PERFORMANCE_TEST) {
            hipEventRecord(stop_iter, 0);  // End timing for this step
            hipEventSynchronize(stop_iter);
            hipEventElapsedTime(&time_step, start_iter, stop_iter);
            printf("Step 2.1 (Distance computation): %f ms\n", time_step);
            hipEventRecord(start_iter);  // Start timing for the next step
        }

        // Assign points to centroids (Step 2.2)
        // 2. 保存舊的質心以便於收斂檢查
        reduce_by_key(
            make_transform_iterator(counting_iterator<int>(0), _1 / opts.n_clusters), // Key input start: Maps indices to clusters by dividing by the number of clusters
            make_transform_iterator(counting_iterator<int>(n_points * opts.n_clusters), _1 / opts.n_clusters), // Key input end: Same mapping for the end of the range
            make_zip_iterator(make_tuple(point_centroid_distances.begin(), counting_iterator<int>(0))), // Value input: Combines the distance values with the point index
            make_discard_iterator(), // Output key: Discards the output keys (not used)
            make_zip_iterator(make_tuple(make_discard_iterator(), point_to_centroid_map.begin())), // Output value: Updates the 'point_to_centroid_map' with the closest centroid index
            equal_to<int>(), // Binary predicate: Uses equality to group keys for reduction
            maximum_by_first() // Binary operation: Chooses the maximum based on the first element of the tuple (distance comparison)
        );

        transform(
            point_to_centroid_map.begin(), // Input start: Beginning of the mapping of points to centroids
            point_to_centroid_map.end(),   // Input end: End of the mapping of points to centroids
            point_to_centroid_map.begin(), // Output: Updates the 'point_to_centroid_map' in place
            _1 % opts.n_clusters           // Unary operation: Takes the modulo of the centroid index with the number of clusters to ensure it remains within valid range
        );

        if (PERFORMANCE_TEST) {
            hipEventRecord(stop_iter, 0);  
            hipEventSynchronize(stop_iter);
            hipEventElapsedTime(&time_step, start_iter, stop_iter);
            printf("Step 2.2 (Assign points to centroids): %f ms\n", time_step);
            hipEventRecord(start_iter); 
        }

        // Zero the centroids and compute new centroids (Step 2.3)
        // 3. 重置新的質心和每個聚類的點數統計
        fill(new_centroids.begin(), new_centroids.end(), 0);

        updateCentroidSums_thrust_op compute_assign_means(opts.dimensions, raw_pointer_cast(point_to_centroid_map.data()), raw_pointer_cast(new_centroids.data()));
        for_each(
            make_zip_iterator(make_tuple(d_points.begin(), counting_iterator<int>(0))),
            make_zip_iterator(make_tuple(d_points.end(), counting_iterator<int>(n_points * opts.dimensions))),
            compute_assign_means
        );

        if (PERFORMANCE_TEST) {
            hipEventRecord(stop_iter, 0);  
            hipEventSynchronize(stop_iter);
            hipEventElapsedTime(&time_step, start_iter, stop_iter);
            printf("Step 2.3 (Compute new centroids): %f ms\n", time_step);
            hipEventRecord(start_iter);  
        }

        d_point_cluster_ids = point_to_centroid_map;

        // Sort point cluster IDs (Step 2.4)
        sort(d_point_cluster_ids.begin(), d_point_cluster_ids.end());

        auto new_end = reduce_by_key(
            d_point_cluster_ids.begin(), // Key input start: Iterator to the beginning of the cluster IDs for each point
            d_point_cluster_ids.end(),   // Key input end: Iterator to the end of the cluster IDs for each point
            make_constant_iterator(1),   // Value input: A constant iterator that generates the value 1 for each key
            d_k_count_keys.begin(),      // Output key: Start of the output range where the unique cluster IDs will be stored
            d_k_counts.begin()           // Output value: Start of the output range where the counts of points per cluster will be stored
        );

        fill(new_end.second, d_k_counts.end(), 0);

        if (PERFORMANCE_TEST) {
            hipEventRecord(stop_iter, 0);  
            hipEventSynchronize(stop_iter);
            hipEventElapsedTime(&time_step, start_iter, stop_iter);
            printf("Step 2.4 (Sort cluster IDs and reduce): %f ms\n", time_step);
            hipEventRecord(start_iter);  
        }

        // Divide centroids by point count (Step 2.5)
        // 5. 計算每個聚類的平均質心
        for (int i = 0; i < opts.dimensions; i++) {
            for_each(
                make_zip_iterator(make_tuple(d_k_count_keys.begin(), d_k_counts.begin())),
                make_zip_iterator(make_tuple(new_end.first, new_end.second)),
                averageLabeledCentroids_thrust_op(opts.dimensions, i, raw_pointer_cast(new_centroids.data()))
            );
        }

        if (PERFORMANCE_TEST) {
            hipEventRecord(stop_iter, 0);  
            hipEventSynchronize(stop_iter);
            hipEventElapsedTime(&time_step, start_iter, stop_iter);
            printf("Step 2.5 (Divide centroids by counts): %f ms\n", time_step);
        }

        // Swap old and new centroids (Step 2.6)
        swap(new_centroids, old_centroids);

        real l1_thresh = opts.threshold / opts.dimensions;

        // Check convergence (Step 2.7)
        // 6. 計算質心的收斂程度
        bool converged = transform_reduce(
            make_zip_iterator(make_tuple(new_centroids.begin(), old_centroids.begin())),
            make_zip_iterator(make_tuple(new_centroids.end(), old_centroids.end())),
            check_convergence_thrust_op(l1_thresh),
            true,
            logical_and<bool>()
        );

        iterations++;
        done = (iterations > opts.max_iterations) || converged;
    }

    // Step 3: Copy final results back to host
    // 計算從設備到主機的傳輸時間 Calculate transfer time from device to host
    hipEventRecord(start_transfer, 0);
    copy(old_centroids.begin(), old_centroids.end(), centroids);
    copy(point_to_centroid_map.begin(), point_to_centroid_map.end(), point_cluster_ids);
    hipEventRecord(stop_transfer, 0);
    hipEventSynchronize(stop_transfer);

    float device_to_host_transfer_time;
    hipEventElapsedTime(&device_to_host_transfer_time, start_transfer, stop_transfer);
    transfer_time += device_to_host_transfer_time;

    // Final timing
    hipEventRecord(stop_total, 0);
    hipEventSynchronize(stop_total);

    // Calculate total runtime
    hipEventElapsedTime(&total_runtime, start_total, stop_total);
    per_iteration_time = total_runtime / iterations;
    float time_taken = total_runtime - transfer_time;

    // Cleanup memory and events
    hipEventDestroy(start_transfer);
    hipEventDestroy(stop_transfer);
    hipEventDestroy(start_total);
    hipEventDestroy(stop_total);
    hipEventDestroy(start_iter);
    hipEventDestroy(stop_iter);

    // Print debug information
    if (opts.debug || PERFORMANCE_TEST) {
        std::cout << "Total runtime: " << total_runtime << " ms" << std::endl;
        std::cout << "Total data transfer time: " << transfer_time << " ms" << std::endl;
        std::cout << "Percent spent in IO: " << transfer_time/total_runtime << " ms" << std::endl;
        std::cout << "Fraction of time spent on data transfer: " << (transfer_time / total_runtime) << std::endl;
        std::cout << "Algorithm Time taken by function: " << time_taken << " ms" << std::endl;
    }

    per_iteration_time = time_taken / iterations;
    print_ExecuteTime(iterations, per_iteration_time, time_taken);
    return iterations;
}